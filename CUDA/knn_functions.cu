#include "hip/hip_runtime.h"
#include "input.h"
#include "knn_functions.h" 
#include <stdlib.h>
#include <stdio.h>
#include <stdint.h>
#include <string.h>
#include <math.h>


//funzione kernel in cui ogni thread computa la distanza tra il proprio sample di test e tutti quelli del train
__global__ void computeDist_kernel(const float* __restrict__ dev_train, const float* __restrict__ dev_test, float* __restrict__ dev_distances){//, int* dev_labels){
	//indice inizio riga
	
	int idx = threadIdx.x+blockDim.x*blockIdx.x;
  	int idy = threadIdx.y+blockDim.y*blockIdx.y;
	
	//printf("cx cy %d %d\n", cx, cy);
	//check extra thread
	if(idx < N && idy <P){
		//printf("cx cy %d %d\n", cx, cy);
		//__shared__ float train[M];
		//trai
		//__shared__ float test[M];
		//__syncthreads();
		float sum = 0.f;
	    #pragma unroll
	    for (int d=0; d<M; ++d) {
	    	//__ldg(d_a + i)
	    	float x = dev_train[idx*M +d];  
	    	float y = dev_test[idy* M +d];
	    	//loat x =__ldg(dev_train + idx*M + d);
	    	//float y =__ldg(dev_test + idy*M + d);
	        float diff = x - y;
	        sum += diff * diff;
	    }
    //return 
		dev_distances[(idy *N) + idx] = sqrtf(sum);//distanceFunction(&dev_train[cy*M], &dev_test[cx*M]);
		//printf("%.2f \n", dev_distances[cx *N + cy]);
		//dev_labels[cx* N + cy] = cy;
	}
}


__global__ void sort_kernel(float* __restrict__ dev_distances, int* __restrict__ dev_labels){
	
	//indice inizio riga
	int index = threadIdx.x + blockDim.x * blockIdx.x;
	//printf(" %d ", index);
	//check extra thread
	if(index < P){
		dev_labels[index*K] = 0;
		#pragma unroll
		for(int i=1; i< N; i++){
			float distanzaCorrente = dev_distances[index*N+i];
        	int indiceCorrente = i;
        	//dev_labels[index*K+i] = i;
        	//printf("distanza corrente %f confronto con %f\n", distanzaCorrente, dev_distances[index*N+ K-1]);
			if( i >= K && distanzaCorrente >= dev_distances[index*N+ K-1]){
            	continue;
        	}
			
			int j = i;
        	if (j > K-1)
            	j = K-1;
        
        	while(j > 0 && dev_distances[index*N+ j-1] > distanzaCorrente){
            	dev_distances[index*N +j] = dev_distances[index*N+j-1];
            	dev_labels[index*K+j] = dev_labels[index*K+j-1];
            	--j;
        	}

        	dev_distances[index*N+j] = distanzaCorrente;
        	dev_labels[index*K+j] = indiceCorrente;	
		}
	}
}



  



