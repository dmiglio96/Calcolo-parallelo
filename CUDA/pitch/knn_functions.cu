#include "hip/hip_runtime.h"
#include "input.h" 
#include "knn_functions.h"
#include <stdlib.h>
#include <stdio.h>
#include <stdint.h>
#include <string.h>
#include <math.h>


//funzione kernel in cui ogni thread computa la distanza tra il proprio sample di test e tutti quelli del train
__global__ void computeDist_kernel(const float *  __restrict__ train, size_t pitchTrain, const float*  __restrict__ test, size_t pitchTest, float*  __restrict__ dist, size_t pitchDist)
{
   int tidx = blockIdx.x*blockDim.x + threadIdx.x;
   int tidy = blockIdx.y*blockDim.y + threadIdx.y;

   if ((tidx < N) && (tidy < P))
   {  
        float sum = 0.f;
        float *row_train = (float *)((char*)train + tidx * pitchTrain);
        float *row_test = (float *)((char*)test + tidy * pitchTest);
        #pragma unroll
        for (int d=0; d<M; ++d) {
          float x = row_train[d];
          float y = row_test[d];
          float diff = x - y;
          sum += diff * diff;
          //printf(" riga test %d riga train %d elemento %d Confronto train %.2f test %.2f \n", tidy, tidx, d, x, y);
        }
      //return 
      float *row_dist = (float *)((char*)dist + tidy * pitchDist);
      row_dist[tidx] = sqrt(sum);
      //dist[(idx *P) + idy] = sqrt(sum);




       /*
       if(tidx == 0 && tidy == 0){
          printf("\nTrain\n");
          for(int i =0; i<N; i++){
            float *row_a = (float *)((char*)train + i * pitchTrain);
            for(int j=0; j< M; j++)
                printf(" %.2f ",row_a[j]);// = row_a[tidx] * tidx * tidy;
            printf("\n");
        }

        printf("\ntest\n");
        for(int i =0; i<P; i++){
            float *row_a = (float *)((char*)test + i * pitchTest);
            for(int j=0; j< M; j++)
                printf(" %.2f ",row_a[j]);// = row_a[tidx] * tidx * tidy;
            printf("\n");
        }

        printf("\n Distanze\n");
        for(int i =0; i<P; i++){
            float *row_a = (float *)((char*)dist + i * pitchDist);
            for(int j=0; j< N; j++)
                printf(" %.2f ",row_a[j]);// = row_a[tidx] * tidx * tidy;
            printf("\n");
        }


       }
       */

       
    }
}


__global__ void sort_kernel(float*  __restrict__ dev_distances, size_t pitchDist, int*  __restrict__ dev_labels, size_t pitchLabel){
	
	//indice inizio riga
	int index = threadIdx.x + blockDim.x * blockIdx.x;
	//printf(" %d ", index);
	//check extra thread
	if(index < P){
		/*if (index == 0){
			printf("\n Distanze\n");
        	for(int i =0; i<P; i++){
            	float *row_a = (float *)((char*)dev_distances + i * pitchDist);
            	for(int j=0; j< N; j++)
                	printf(" %.2f ",row_a[j]);// = row_a[tidx] * tidx * tidy;
            	printf("\n");
        	}
		}
		*/
		int *row_label = (int *)((char*)dev_labels + index * pitchLabel);
		row_label[0] = 0;
		#pragma unroll
		for(int i=1; i< N; i++){
			int *row_label_loop = (int *)((char*)dev_labels + index * pitchLabel);
			
			float *row_dist = (float *)((char*)dev_distances + index * pitchDist);
			
			float distanzaCorrente = row_dist[i];
        	int indiceCorrente = i;
        	//dev_labels[index*K+i] = i;
        	//printf("distanza corrente %f confronto con %f\n", distanzaCorrente, dev_distances[index*N+ K-1]);
			if( i >= K && distanzaCorrente >= row_dist[K-1]){
            	continue;
        	}
			
			int j = i;
        	if (j > K-1)
            	j = K-1;
        
        	while(j > 0 && row_dist[j-1] > distanzaCorrente){
            	row_dist[j] = row_dist[j-1];
            	row_label_loop[j] = row_label_loop[j-1];
            	--j;
        	}

        	row_dist[j] = distanzaCorrente;
        	row_label_loop[j] = indiceCorrente;	
		}

		/*if(index == 0){
			if (index == 0){
			printf("\n Distanze dopo\n");
        	for(int i =0; i<P; i++){
            	float *row_a = (float *)((char*)dev_distances + i * pitchDist);
            	for(int j=0; j< K; j++)
                	printf(" %.2f ",row_a[j]);// = row_a[tidx] * tidx * tidy;
            	printf("\n");
        	}

        	printf("\n label \n");
        	for(int i =0; i<P; i++){
            	int *row_a = (int *)((char*)dev_labels + i * pitchLabel);
            	for(int j=0; j< K; j++)
                	printf(" %d ",row_a[j]);// = row_a[tidx] * tidx * tidy;
            	printf("\n");
        	}
		}

		}
		*/

	}
}



  



