#include "hip/hip_runtime.h"
#include "input.h" 
#include "knn_functions.h"
#include <stdlib.h>
#include <stdio.h>
#include <stdint.h>
#include <string.h>
#include <math.h>

__global__ void singleDistance(const float* __restrict__ dev_train, const float* __restrict__ dev_test, float* __restrict__ distance){
	int id = threadIdx.x; 
	float local_distance = dev_train[id] - dev_test[id];
	local_distance = local_distance *local_distance;
	atomicAdd(distance, local_distance);
	//distance = distance +local_distance
}

//funzione kernel in cui ogni thread computa la distanza tra il proprio sample di test e tutti quelli del train
__global__ void computeDist_kernel(const float* __restrict__ dev_train, const float* __restrict__ dev_test, float* __restrict__ dev_distances){//, int* dev_labels){
	//indice inizio riga
	
	int idx = threadIdx.x+blockDim.x*blockIdx.x;
  	int idy = threadIdx.y+blockDim.y*blockIdx.y;
	
	//printf("cx cy %d %d\n", cx, cy);
	//check extra thread
	if(idx < N && idy <P){
	    singleDistance<<<1, M>>>(&dev_train[idx*M], &dev_test[idy* M], &dev_distances[(idy *N) + idx]);
		hipDeviceSynchronize();
		dev_distances[(idy *N) + idx] = sqrtf(dev_distances[(idy *N) + idx]);
	}
}


__global__ void sort_kernel(float* __restrict__ dev_distances, int* __restrict__ dev_labels){
	
	//indice inizio riga
	int index = threadIdx.x + blockDim.x * blockIdx.x;
	//printf(" %d ", index);
	//check extra thread
	if(index < P){
		dev_labels[index*K] = 0;
		#pragma unroll
		for(int i=1; i< N; i++){
			float distanzaCorrente = dev_distances[index*N+i];
        	int indiceCorrente = i;
        	//dev_labels[index*K+i] = i;
        	//printf("distanza corrente %f confronto con %f\n", distanzaCorrente, dev_distances[index*N+ K-1]);
			if( i >= K && distanzaCorrente >= dev_distances[index*N+ K-1]){
            	continue;
        	}
			
			int j = i;
        	if (j > K-1)
            	j = K-1;
        
        	while(j > 0 && dev_distances[index*N+ j-1] > distanzaCorrente){
            	dev_distances[index*N +j] = dev_distances[index*N+j-1];
            	dev_labels[index*K+j] = dev_labels[index*K+j-1];
            	--j;
        	}

        	dev_distances[index*N+j] = distanzaCorrente;
        	dev_labels[index*K+j] = indiceCorrente;	
		}
	}
}



  



