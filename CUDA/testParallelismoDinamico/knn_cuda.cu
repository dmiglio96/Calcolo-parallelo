#include <stdint.h>
#include <stdio.h> 
#include <stdlib.h>
#include <string.h>
#include <math.h>

#include "input.h"
#include "knn_functions.h"
#include "check.h"
#include "hipError_t.h"
#include "utility.h"


int main(int argc, char* argv[]) {
	//salvare o meno il risultato su file
    bool saveData = true;
	bool checkresult = false;
	const char * trainFile = argv[1];
	const char * testFile = argv[2];
	
	//device
	int deviceIndex = 0;

	if((argc -1) != 2){
		printf("Errore non sono stati specificati correttamente i file del dataset!\n");
		exit(EXIT_FAILURE);
	}

	if (K > N){
		printf("Errore il numero di vicini non può essere superiore al numero di sample!\n");
		exit(EXIT_FAILURE);
	}

	if (K % 2 == 0){
		printf("Inserire un numero di vicini dispari!\n");
		exit(EXIT_FAILURE);
	}


	//numero di schede presenti
	int count;
	HANDLE_ERROR( hipGetDeviceCount( &count ) );
    
    //check esistenza scheda disponbile
    if(deviceIndex < count)
    {
        HANDLE_ERROR(hipSetDevice(deviceIndex));
    }
    else
    {
        printf("Device non disponbile!\n");
        exit(EXIT_FAILURE);        
    }

    // proprietà della scheda video
    hipDeviceProp_t prop;
    HANDLE_ERROR(hipGetDeviceProperties(&prop, deviceIndex));

    //printf("M : %d Max threads per block: %d\n",M, prop.maxThreadsPerBlock );
	//printf("Max thread dimensions: (%d, %d, %d)\n",prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2] );
	//printf("Max grid dimensions: (%d, %d, %d)\n",prop.maxGridSize[0], prop.maxGridSize[1],prop.maxGridSize[2] );
	//printf("\n" );
    /*
    int maxthread;
    hipDeviceGetAttribute(&maxthread, maxThreadsPerBlock);
    //Check sforamento numero di thread per blocco 
    if (BLOCK_SIZE * BLOCK_SIZE > maxthread){
    	printf("Errore, superato massimo numero di thread per blocco!\n");
    	exit(EXIT_FAILURE);
    }
    */


	// misurare il tempo di esecuzione
	hipEvent_t start, stop, stopRead, stopSendData, primoStep, secondoStep;
	HANDLE_ERROR( hipEventCreate( &start ) );
	HANDLE_ERROR( hipEventCreate( &stop ) );
	HANDLE_ERROR( hipEventCreate( &stopRead ) );
	HANDLE_ERROR( hipEventCreate( &stopSendData ) );
	HANDLE_ERROR( hipEventCreate( &primoStep ) );
	HANDLE_ERROR( hipEventCreate( &secondoStep ) );
	
	HANDLE_ERROR( hipEventRecord( start, 0 ) );

	float * trainingData= (float *) malloc(N* M * sizeof(float));
	float * testingData= (float *) malloc(P* M * sizeof(float));
	//HANDLE_ERROR( hipHostAlloc( (void**)&trainingData, N*M * sizeof( *trainingData ), hipHostMallocDefault ) );
	//HANDLE_ERROR( hipHostAlloc( (void**)&testingData, P*M * sizeof( *testingData ), hipHostMallocDefault ) );

	int * classesTraining = (int*) malloc(N *sizeof(int));
	int * classesTesting = (int*)  malloc(P *sizeof(int));

	float * dist = (float *) malloc(P* N * sizeof(float));
	//HANDLE_ERROR( hipHostAlloc( (void**)&dist, P*M * sizeof( *dist ), hipHostMallocDefault ) );
	
	
	if(trainingData == NULL || testingData == NULL || classesTesting == NULL || classesTraining == NULL){
		printf("Not enough memory!\n");
		exit(EXIT_FAILURE);
	}

	
	read_file(trainFile, N, M, trainingData, classesTraining);
	read_file(testFile, P, M, testingData, classesTesting);

	printf("nome file %s \n", trainFile);
	printf("nome file test %s \n", testFile);

	// get stop time, and display the timing results
	HANDLE_ERROR( hipEventRecord( stopRead, 0 ) );
	HANDLE_ERROR( hipEventSynchronize( stopRead ) );
	
	float elapsedTimeRead;
	HANDLE_ERROR( hipEventElapsedTime( &elapsedTimeRead, start, stopRead ) );
	printf( "Lettura dati eseguita in: %f \n", elapsedTimeRead/1000 );
  	
	// puntattori ai dati sul device
	float* dev_train; 
	
	float* dev_test;

	float* dev_dist;

	int* dev_label;
	
	
	// alloco memoria per il dataset sulla gpu in memoria globale
	HANDLE_ERROR( hipMalloc( (void**)&dev_train, N * M * sizeof(float) ) );
	
	HANDLE_ERROR( hipMalloc( (void**)&dev_test, P * M * sizeof(float) ) );

	//allocco matrice distanze e relative label
	HANDLE_ERROR( hipMalloc( (void**)&dev_dist, P* N  * sizeof(float) ) );

	//HANDLE_ERROR( hipMalloc( (void**)&dev_label, P * N * sizeof(int) ) );

	
	
	// copia elementi del dataset
	HANDLE_ERROR( hipMemcpy( dev_train, trainingData, N * M * sizeof(float), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy( dev_test, testingData, P * M * sizeof(float), hipMemcpyHostToDevice ) );	
	//HANDLE_ERROR( hipMemcpy( dev_dist, dist, N * P * sizeof(float), hipMemcpyHostToDevice ) );

	HANDLE_ERROR( hipEventRecord( stopSendData, 0 ) );
	HANDLE_ERROR( hipEventSynchronize( stopSendData ) );
	
	HANDLE_ERROR( hipEventElapsedTime( &elapsedTimeRead, start, stopSendData ) );
	printf( "Copia dati su GPU eseguita dopo : %f  secondi\n", elapsedTimeRead/1000 );
	
	//HANDLE_ERROR( hipMemcpy( dev_label, label, N * P * sizeof(int), hipMemcpyHostToDevice ) );

	// creo blocchi da BLOCK_SIZE * BLOCK_SIZE thread
	dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1); 

	//Numero di blocchi
	int dim_row = (P +1 % BLOCK_SIZE == 0) ? P / BLOCK_SIZE : P / BLOCK_SIZE + 1;
	int dim_col = (N + 1 % BLOCK_SIZE == 0) ? N / BLOCK_SIZE : N / BLOCK_SIZE + 1;
	

	dim3 grid(dim_col, dim_row, 1); // a grid of CUDA thread blocks
	//printf("Numero di blocchi %d %d da %d \n", dim_row, dim_col, BLOCK_SIZE); 
	
	//hipFuncSetCacheConfig(reinterpret_cast<const void*>(computeDist_kernel), hipFuncCachePreferL1);
	// calcola distanza euclidea tra punti train e test
	computeDist_kernel<<<grid, block>>>(dev_train, dev_test, dev_dist);//, dev_label);

	int * label = (int*) malloc(P * K *sizeof(int));
	int* countsLabel = (int*) malloc(sizeof(int)* LABELS);
	int* confusionMatrix = (int*) malloc(sizeof(int)* LABELS * LABELS);

	if(confusionMatrix ==NULL || countsLabel == NULL || label == NULL){
		printf("Not enough memory!\n");
		exit(EXIT_FAILURE);
	}

	// inizializza a zero la matrice di confusione
	initilizeArray(confusionMatrix, LABELS*LABELS, 0);
	
	// barriera per assicurarsi che tutte le distanze siano state calcolate
	hipDeviceSynchronize();
	HANDLE_ERROR( hipEventRecord(  primoStep, 0 ) );
	HANDLE_ERROR( hipEventSynchronize(  primoStep ) );
	
	HANDLE_ERROR( hipEventElapsedTime( &elapsedTimeRead, start, primoStep ) );
	printf( "Distanze calcolate dopo : %f  secondi\n", elapsedTimeRead/1000 );
	

	//dealloco dataset su device non più utile
	
	HANDLE_ERROR( hipFree(dev_train) );
    HANDLE_ERROR( hipFree(dev_test) );
	
	//hipDeviceSynchronize();	
	HANDLE_ERROR( hipMalloc( (void**)&dev_label, P * K * sizeof(int) ) );
	//HANDLE_ERROR( hipMemcpy( dev_label, label, P*K * sizeof(int), hipMemcpyHostToDevice ) );

	dim3 blockSort(BLOCK_SIZE, 1, 1);
	dim3 gridSort(dim_row, 1, 1);
	//printf("Numero di blocchi per il sort %d da %d \n", dim_row, BLOCK_SIZE); 
	sort_kernel<<<gridSort, blockSort>>>(dev_dist, dev_label);
	// barriera per assicurare che siano tutti ordinat

	hipDeviceSynchronize();

	//recupero risultati dalla GPU
	//HANDLE_ERROR(hipMemcpy(dist , dev_dist, P * N * sizeof(float), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR(hipMemcpy(label , dev_label, P * K * sizeof(int), hipMemcpyDeviceToHost ) );
	
	HANDLE_ERROR( hipEventRecord(  secondoStep, 0 ) );
	HANDLE_ERROR( hipEventSynchronize(  secondoStep ) );
	
	HANDLE_ERROR( hipEventElapsedTime( &elapsedTimeRead, start, secondoStep ) );
	
	printf( "Ordinate e ricevute dopo : %f  secondi\n", elapsedTimeRead/1000 );
	
	/*printf("Dopoooooooo\n");
	for(int i=0; i < P; i++){
		for(int j=0; j < K; j++)
			printf(" %d ", label[i*K +j]);
		printf("\n\n");
	}
	*/
	
	
	
	// numero di errori di classificazione commessi dall'algoritmo KNN
	int error = 0;
	
	//il calcolo della matrice di confusione finale viene lasciato alla cpu
	for (int i=0; i<P; i++){
		initilizeArray(countsLabel, LABELS, 0);
		int bestLabel = 0;
		for(int j=0; j<K; j++){	
			int indice = label[i*K+j];
			int classe = classesTraining[indice]; 
			countsLabel[classe] = countsLabel[classe] + 1;
			if(countsLabel[classe] > countsLabel[bestLabel])
				bestLabel = classe;
			}

		int realLabel = classesTesting[i];
		if (realLabel != bestLabel){
			error = error + 1;
		}
			
		//update confusion matrix
		confusionMatrix[realLabel * LABELS + bestLabel] = confusionMatrix[realLabel * LABELS + bestLabel] +1;	
	}
	
	//stampa Confusion matrix
	//printConfusionMatrix(confusionMatrix);
	//printf("Errori totali: %d\n", error);
	//printf("Record corretti: %d accuratezza (%.2f%%); ", P - error, 100 - ((float) error / P) * 100);
	
	// controllo risultato con il seriale
	if(checkresult == true){
		checkResultKNN(trainingData, testingData, classesTraining, classesTesting, confusionMatrix);
	}	
	
	// dealloca memoria CPU

	//HANDLE_ERROR( hipHostFree( trainingData) );
	//HANDLE_ERROR( hipHostFree( testingData ) );
	//HANDLE_ERROR( hipHostFree( dist ) );
	
	free(trainingData); trainingData = NULL;
	free(testingData); testingData = NULL;
	free(dist); dist=NULL;
	
	free(classesTraining); classesTraining = NULL;
	free(classesTesting); classesTesting = NULL;
	
	free(confusionMatrix); confusionMatrix=NULL;
	
	free(label); label=NULL;
	free(countsLabel); countsLabel= NULL;
	
	//dealloco memoria GPU
    //HANDLE_ERROR( hipFree(dev_train) );
    //HANDLE_ERROR( hipFree(dev_test) );

    HANDLE_ERROR( hipFree(dev_label ) );
    HANDLE_ERROR( hipFree(dev_dist ) );
    	
	
	// conteggio tempo totale di esecuzione
	HANDLE_ERROR( hipEventRecord( stop, 0 ) );
	HANDLE_ERROR( hipEventSynchronize( stop ) );
	
	float elapsedTime;
	HANDLE_ERROR( hipEventElapsedTime( &elapsedTime, start, stop ) );
	
	printf( "Total time: %f \n", elapsedTime/1000 );
	HANDLE_ERROR( hipEventDestroy( start ) );
	HANDLE_ERROR( hipEventDestroy( stop ) );
	//HANDLE_ERROR( hipEventDestroy( stopRead ) );

	//save on file
	if(saveData == true)
      saveResultsOnFile(elapsedTime/1000);

	return 0;
}
